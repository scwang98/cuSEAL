
#include "kernelprovider.h"

namespace sigma {

    bool KernelProvider::initialized = false;

    void KernelProvider::checkInitialized() {
        if (!initialized)
            throw std::invalid_argument("KernelProvider not initialized.");
    }

    void KernelProvider::initialize() {
        hipSetDevice(0);
        initialized = true;
    }

    template<typename T>
    T *KernelProvider::malloc(size_t length) {
        checkInitialized();
        if (length == 0) return nullptr;
        T *ret;
        auto status = hipMalloc((void **) &ret, length * sizeof(T));
        if (status != hipSuccess)
            throw std::runtime_error("Cuda Malloc failed.");
        return ret;
    }

    template<typename T>
    void KernelProvider::free(T *pointer) {
        checkInitialized();
        hipFree(pointer);
    }

    template<typename T>
    void KernelProvider::copy(T *deviceDestPtr, const T *hostFromPtr, size_t length) {
        checkInitialized();
        if (length == 0) return;
        auto status = hipMemcpy(deviceDestPtr, hostFromPtr, length * sizeof(T), hipMemcpyHostToDevice);
        if (status != hipSuccess)
            throw std::runtime_error("Cuda copy from host to device failed.");
    }

    template<typename T>
    void KernelProvider::copyOnDevice(T *deviceDestPtr, const T *deviceFromPtr, size_t length) {
        checkInitialized();
        if (length == 0) return;
        auto status = hipMemcpy(deviceDestPtr, deviceFromPtr, length * sizeof(T), hipMemcpyDeviceToDevice);
        if (status != hipSuccess)
            throw std::runtime_error("Cuda copy on device failed.");
    }

    template<typename T>
    void KernelProvider::retrieve(T *hostDestPtr, const T *deviceFromPtr, size_t length) {
        checkInitialized();
        if (length == 0) return;
        auto status = hipMemcpy(hostDestPtr, deviceFromPtr, length * sizeof(T), hipMemcpyDeviceToHost);
        if (status != hipSuccess)
            throw std::runtime_error("Cuda retrieve from device to host failed.");
    }

    template<typename T>
    void KernelProvider::memsetZero(T *devicePtr, size_t length) {
        if (length == 0) return;
        hipMemset(devicePtr, 0, sizeof(T) * length);
    }
}