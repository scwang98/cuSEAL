#include "hip/hip_runtime.h"
//
// Created by scwang on 2024/3/6.
//

#include "calculate.h"
#include <iostream>
#include <fstream>
#include <utility>
#include <vector>
#include <unordered_set>
#include <sigma.h>
#include <iomanip>

#include "extern/jsoncpp/json/json.h"
#include "util/configmanager.h"
#include "util/vectorutil.h"
#include "util/keyutil.h"
#include "util/safequeue.h"

using namespace std;

#define DIMENSION 512
#define THREAD_SIZE 8
#define PROBE_SIZE 10

std::string gallery_data_path(const std::string &directory, size_t index) {
    std::ostringstream oss;
    oss << std::setw(5) << std::setfill('0') << index;
    return directory + "/gallery_data/gallery_" + oss.str() + "_results.dat";
}

std::string ip_results_path(const std::string &directory, size_t index) {
    return directory + "/probe_" + std::to_string(index) + "_results.dat";
}

void save_results(vector<vector<vector<sigma::Ciphertext>>> &results, std::ofstream &ofs) {
    size_t size1 = results.size();
    ofs.write(reinterpret_cast<const char *>(&size1), sizeof(size_t));
    for (auto &result : results) {
        size_t size2 = result.size();
        ofs.write(reinterpret_cast<const char *>(&size2), sizeof(size_t));
        for (auto &result1: result) {
            size_t size3 = result1.size();
            ofs.write(reinterpret_cast<const char *>(&size3), sizeof(size_t));
            for (auto &result2: result1) {
                result2.save(ofs);
            }
        }
    }
}

void save_cluster_indexes(vector<vector<size_t>> &cluster_indexes, std::ofstream &ofs) {
    size_t size1 = cluster_indexes.size();
    ofs.write(reinterpret_cast<const char *>(&size1), sizeof(size_t));
    for (auto &index : cluster_indexes) {
        size_t size2 = index.size();
        ofs.write(reinterpret_cast<const char *>(&size2), sizeof(size_t));
        ofs.write(reinterpret_cast<const char *>(index.data()), size2 * sizeof(int64_t));
    }
}

class Task {

public:

    std::vector<float> probe_data;
    std::vector<size_t> indexes;
    size_t finished_part;
    int probe_index;
    sigma::Ciphertext c1_sum;
    vector<vector<vector<sigma::Ciphertext>>> results;
    vector<vector<size_t>> cluster_indexes;

    Task() = default;

    Task(const std::vector<float>& data, std::vector<size_t> &indexes, int gpu_count, int probe_index) : probe_data(data), indexes(indexes), probe_index(probe_index) {
        finished_part = 0;
        results.resize(gpu_count);
    }

};

struct IPIndex {
    float inner_product;
    size_t index;

    IPIndex(float inner_product, size_t index) : inner_product(inner_product), index(index) {}

    bool operator<(const IPIndex& other) const {
        return inner_product > other.inner_product;
    }
};

class TaskManager {

    std::vector<float> centroids;

public:
    int gpu_count;
    unordered_set<Task *> set;
    vector<util::safe_queue<Task *, 100>> queues;

    util::safe_queue<Task *, 400> finished_queue;

    TaskManager(int gpu_count, std::vector<float> &centroids) : gpu_count(gpu_count), centroids(centroids) {
        queues.resize(gpu_count);
    }

    Task *start_task(const std::vector<float>& data, int probe_index) {
        auto size = centroids.size() / DIMENSION;
        std::priority_queue<IPIndex> pq;
        for (int i = 0; i < size; ++i) {
            auto start = centroids.data() + DIMENSION * i;
            float ip = 0;
            for (int j = 0; j < DIMENSION; ++j) {
                ip += (*(start + j) * data[j]);
            }
            pq.emplace(ip, i);
            if (i >= 5) {
                pq.pop();
            }
        }
        std::vector<size_t> indexes;
        for (int i = 0; i < pq.size(); ++i) {
            indexes.push_back(pq.top().index);
            pq.pop();
        }

        auto task = new Task(data, indexes, gpu_count, probe_index);
        set.insert(task);
        for (uint i = 0; i < gpu_count; i++) {
            queues[i].push(task);
        }
        return task;
    }

    void probe_finished() {
        auto task = new Task();
        for (uint i = 0; i < gpu_count; i++) {
            queues[i].push(task);
        }
    }

    void task_finished(Task *task) {
        finished_queue.push(task);
    }
};

vector<vector<vector<sigma::Ciphertext>>> gallery_data_cluster;
std::vector<std::vector<int64_t>> indexes;
std::vector<std::vector<float>> probe_data;

TaskManager *task_manager;

void calculate_thread(int gpu_index, int gpu_count, int cluster_per_gpu, sigma::SIGMAContext &context, const sigma::Ciphertext &c1, double scale) {
    hipSetDevice(gpu_index);

    auto gpu_gallery_data = gallery_data_cluster[gpu_index];

    const uint cluster_index_start = cluster_per_gpu * gpu_index;
    const uint cluster_index_end = cluster_index_start + gpu_gallery_data.size();

    sigma::CKKSEncoder encoder(context);
    sigma::Evaluator evaluator(context);

    sigma::Ciphertext c1_sum;
    sigma::Ciphertext c1_row;
    sigma::Ciphertext result;
    sigma::Ciphertext row;

    std::vector<sigma::Plaintext> encoded_probes(DIMENSION);

    while (true) {

        auto task = task_manager->queues[gpu_index].pop();

        const auto &probe = task->probe_data;

        if (probe.empty()) {
            // TODO: stop
            task_manager->task_finished(task);
            break;
        }

        encoder.cu_encode(probe[0], scale, encoded_probes[0]);
        for (int i = 1; i < DIMENSION; ++i) {
            encoder.cu_encode(probe[i], scale, encoded_probes[i]);
        }

        if (task->probe_index % gpu_count == gpu_index) {
            evaluator.cu_multiply_plain(c1, encoded_probes[0], c1_sum);
            for (int i = 1; i < DIMENSION; ++i) {
                evaluator.cu_multiply_plain(c1, encoded_probes[i], c1_row);
                evaluator.cu_add_inplace(c1_sum, c1_row);
            }
            c1_sum.retrieve_to_host();
            task->c1_sum.copy_from(c1_sum, false);
        }

        vector<size_t> filtered_indexes;
        for (auto index : task->indexes) {
            if (cluster_index_start <= index && index < cluster_index_end) {
                filtered_indexes.push_back(index - cluster_index_start);
            }
        }

        task->results[gpu_index].resize(filtered_indexes.size());

        for (int cluster_index = 0; cluster_index < filtered_indexes.size(); cluster_index++) {
            auto index = filtered_indexes[cluster_index];
            auto gallery_data = gpu_gallery_data[index];
            for (size_t offset = 0; offset < gallery_data.size(); offset += DIMENSION) {

                evaluator.cu_multiply_plain(gallery_data[offset], encoded_probes[0], result);
                for (size_t i = 1; i < DIMENSION; i++) {
                    evaluator.cu_multiply_plain(gallery_data[offset + i], encoded_probes[i], row);
                    evaluator.cu_add_inplace(result, row);
                }

                result.retrieve_to_host();
                task->results[gpu_index][cluster_index].emplace_back(result, false);
                task->cluster_indexes[gpu_index].push_back(index + cluster_index_start);
            }
        }

        task_manager->task_finished(task);
    }
}

void task_for_gpu(int gpu_index, int gpu_count, int cluster_per_gpu, sigma::SIGMAContext &context, const sigma::Ciphertext &origin_c1, double scale) {
    hipSetDevice(gpu_index);
    auto &gallery_data = gallery_data_cluster[gpu_index];
    for (auto &cluster_data : gallery_data) {
        cout << cluster_data.size() << endl;
        cout << "wsccccc" << endl;
        for (auto &ciphertext : cluster_data) {
            ciphertext.copy_to_device();
        }
    }

    sigma::Ciphertext c1 = origin_c1;
    c1.copy_to_device();

    vector<thread> threads;
    for (int i = 0; i < THREAD_SIZE; i++) {
        threads.emplace_back(calculate_thread, gpu_index, gpu_count, cluster_per_gpu, std::ref(context), std::ref(c1), std::ref(scale));
    }

    for (auto &thread: threads) {
        if (thread.joinable()) {
            thread.join();
        }
    }
}

void save_thread(const std::string &result_directory) {
    while (true) {
        auto task = task_manager->finished_queue.pop();
        task->finished_part++;
        if (task->finished_part < task_manager->gpu_count) {
            continue;
        }
        if (task->probe_data.empty()) {
            break;
        }

        std::ofstream ofs(ip_results_path(result_directory, task->probe_index), std::ios::binary);
        task->c1_sum.save(ofs);
        save_results(task->results, ofs);
        save_cluster_indexes(task->cluster_indexes, ofs);

        // TODO: 数据存储
        task_manager->set.erase(task);
        delete task;
    }
}

void calculate(const std::string &probe_path, const std::string &encrypted_directory, const std::string &result_directory) {

    int gpu_count = 0;
    hipGetDeviceCount(&gpu_count);

    std::ifstream indexes_ifs("../data/gallery_data/gallery_indexes.dat", std::ios::binary);
    size_t cluster_size = 0;
    indexes_ifs.read(reinterpret_cast<char*>(&cluster_size), sizeof(size_t));

    std::ifstream centroids_ifs("../data/gallery_data/gallery_centroids.dat");
    std::vector<float> centroids(cluster_size * DIMENSION);
    indexes_ifs.read(reinterpret_cast<char*>(centroids.data()), cluster_size * DIMENSION * sizeof(float));
    centroids_ifs.close();

    size_t cluster_per_gpu = cluster_size / gpu_count;
    if (cluster_size % gpu_count != 0) {
        cluster_per_gpu++;
    }

    size_t poly_modulus_degree = ConfigUtil.int64ValueForKey("poly_modulus_degree");
    size_t scale_power = ConfigUtil.int64ValueForKey("scale_power");
    double scale = pow(2.0, scale_power);

    task_manager = new TaskManager(gpu_count, centroids);

    sigma::KernelProvider::initialize();

    sigma::EncryptionParameters params(sigma::scheme_type::ckks);
    params.set_poly_modulus_degree(poly_modulus_degree);
    auto modulus_bit_sizes = ConfigUtil.intVectorValueForKey("modulus_bit_sizes");
    params.set_coeff_modulus(sigma::CoeffModulus::Create(poly_modulus_degree, modulus_bit_sizes));
    sigma::SIGMAContext context(params);

    std::string encrypted_c1_data_path = encrypted_directory;
    if (encrypted_directory.back() != '/') {
        encrypted_c1_data_path += "/";
    }
    encrypted_c1_data_path += "encrypted_c1.dat";
    std::ifstream c1_ifs(encrypted_c1_data_path, std::ios::binary);
    sigma::Ciphertext c1;
    c1.use_half_data() = true;

    c1.load(context, c1_ifs);
    c1_ifs.close();

    gallery_data_cluster.resize(gpu_count);
    for (uint gpu_index = 0; gpu_index < gpu_count - 1; gpu_index++) {
        gallery_data_cluster[gpu_index].resize(cluster_per_gpu);
    }
    gallery_data_cluster[gpu_count - 1].resize(cluster_size - (cluster_per_gpu * (gpu_count - 1)));
    indexes.resize(cluster_size);
    for (uint cluster_idx = 0; cluster_idx < cluster_size; cluster_idx++) {
        size_t indexes_size = 0;
        indexes_ifs.read(reinterpret_cast<char*>(&indexes_size), sizeof(size_t));
        indexes[cluster_idx].resize(indexes_size);
        indexes_ifs.read(reinterpret_cast<char*>(indexes[cluster_idx].data()), indexes_size * sizeof(int64_t));

        auto gpu_idx = cluster_idx / cluster_per_gpu;
        auto idx = cluster_idx % cluster_per_gpu;

        std::ifstream cluster_ifs(gallery_data_path(encrypted_directory, cluster_idx), std::ios::binary);
        size_t gallery_size = 0;
        cluster_ifs.read(reinterpret_cast<char*>(&gallery_size), sizeof(size_t));
        auto &cluster = gallery_data_cluster[gpu_idx][idx];
        cluster.resize(gallery_size);
        for (auto &ciphertext : cluster) {
            ciphertext.use_half_data() = true;
            ciphertext.load(context, cluster_ifs);
        }
    }

    vector<thread> threads;
    for (int i = 0; i < gpu_count; i++) {
        threads.emplace_back(task_for_gpu, i, gpu_count, cluster_per_gpu, std::ref(context), std::ref(c1), scale);
    }

    thread save_thread_ptr(save_thread, std::ref(result_directory));

    probe_data = util::read_npy_data(probe_path);


    for (int i = 0; i < probe_data.size(); i++) {
        auto &data = probe_data[i];
        task_manager->start_task(data, i);
    }

    task_manager->probe_finished();

    for (auto &thread: threads) {
        if (thread.joinable()) {
            thread.join();
        }
    }

    if (save_thread_ptr.joinable()) {
        save_thread_ptr.join();
    }

    probe_data.clear();

}